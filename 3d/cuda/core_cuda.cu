#include "hip/hip_runtime.h"
// Main solver routines for heat equation solver
#include "heat.hpp"
#include <hip/hip_runtime.h>
#include "error_checks.h"

// Update the temperature values using five-point stencil */
__global__ void evolve_kernel(double *currdata, double *prevdata, double a, double dt, int nx, int ny, int nz,
                       double inv_dx2, double inv_dy2, double inv_dz2)
{

    // CUDA threads are arranged in column major order; thus k index from x, j from y, ...
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.z * blockDim.z + threadIdx.z;

    if (i > 0 && j > 0 && k > 0 && i < nx+1 && j < ny+1 && k < nz+1) {
      int ind = i * (ny + 2) * (nz + 2) + j * (nz + 2) + k;
      int ip = (i + 1) * (ny + 2) * (nz + 2) + j * (nz + 2) + k;
      int im = (i - 1) * (ny + 2) * (nz + 2) + j * (nz + 2) + k;
      int jp = i * (ny + 2) * (nz + 2) + (j + 1) * (nz + 2) + k;
      int jm = i * (ny + 2) * (nz + 2) + (j - 1) * (nz + 2) + k;
      int kp = i * (ny + 2) * (nz + 2) + j * (nz + 2) + (k + 1);
      int km = i * (ny + 2) * (nz + 2) + j * (nz + 2) + (k - 1);
      currdata[ind] = prevdata[ind] + a * dt * (
                  ( prevdata[ip] - 2.0 * prevdata[ind] + prevdata[im] ) * inv_dx2 +
                  ( prevdata[jp] - 2.0 * prevdata[ind] + prevdata[jm] ) * inv_dy2 +
                  ( prevdata[kp] - 2.0 * prevdata[ind] + prevdata[km] ) * inv_dz2
      );
    }
}

void evolve(Field& curr, Field& prev, const double a, const double dt)
{

  int nx = curr.nx;
  int ny = curr.ny;
  int nz = curr.nz;

  auto inv_dx2 = 1.0 / (prev.dx * prev.dx);
  auto inv_dy2 = 1.0 / (prev.dy * prev.dy);
  auto inv_dz2 = 1.0 / (prev.dz * prev.dz);
  
  // CUDA thread settings 
  const int blocksize = 8;  //!< CUDA thread block dimension
  dim3 dimBlock(blocksize, blocksize, blocksize);
  // CUDA threads are arranged in column major order; thus make ny x nx grid
  dim3 dimGrid((nz + 2 + blocksize - 1) / blocksize,
               (ny + 2 + blocksize - 1) / blocksize,
               (nx + 2 + blocksize - 1) / blocksize);

  auto currdata = curr.temperature_dev;
  auto prevdata = prev.temperature_dev;

  evolve_kernel<<<dimGrid, dimBlock>>>(currdata, prevdata, a, dt, nx, ny, nz, 
                                         inv_dx2, inv_dy2, inv_dz2);
  hipDeviceSynchronize();

}

#ifndef CUDA_MANAGED
void enter_data(Field& field1, Field& field2)
{
    size_t field_size = (field1.nx + 2) * (field1.ny + 2) * (field1.nz + 2) * sizeof(double);

    CUDA_CHECK( hipMalloc(&field1.temperature_dev, field_size) ); 
    CUDA_CHECK( hipMalloc(&field2.temperature_dev, field_size) );

    CUDA_CHECK( hipMemcpy(field1.temperature_dev, field1.temperature.data(), field_size, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(field2.temperature_dev, field2.temperature.data(), field_size, hipMemcpyHostToDevice) );
}

void exit_data(Field& field1, Field& field2)
{
    size_t field_size = (field1.nx + 2) * (field1.ny + 2) * (field1.nz + 2) * sizeof(double);

    hipMemcpy(field1.temperature.data(), field1.temperature_dev, field_size, hipMemcpyDeviceToHost) ;
    // CHECK_ERROR_MSG("WTF");
    CUDA_CHECK( hipMemcpy(field2.temperature.data(), field2.temperature_dev, field_size, hipMemcpyDeviceToHost) );

    // CUDA_CHECK( hipFree(field1.temperature_dev) );
    // CUDA_CHECK( hipFree(field2.temperature_dev) );
}

/* Copy a temperature field from the device to the host */
void update_host(Field& field)
{
    size_t field_size = (field.nx + 2) * (field.ny + 2) * (field.nz + 2) * sizeof(double);

    CUDA_CHECK( hipMemcpy(field.temperature.data(), field.temperature_dev, field_size, hipMemcpyDeviceToHost) );
}
/* Copy a temperature field from the host to the device */
void update_device(Field& field)
{
    size_t field_size = (field.nx + 2) * (field.ny + 2) * (field.nz + 2) * sizeof(double);
    hipMemcpy(field.temperature_dev, field.temperature.data(), field_size, hipMemcpyHostToDevice);
}
#endif

